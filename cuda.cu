#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <random>
#include <eigen/Sparse>
using namespace std;
using namespace Eigen;

#define DATA_TYPE int
#define WIDTH_TILE 32

typedef struct Matrix_multiplication
{
	Matrix_multiplication(): ROW_X(1024), COL_X(1024), ROW_Y(1024), COL_Y(1024) {initialize();}
	Matrix_multiplication(int input_ROW_X, int input_COL_X, int input_ROW_Y, int input_COL_Y): ROW_X(input_ROW_X), COL_X(input_COL_X), ROW_Y(input_ROW_Y), COL_Y(input_COL_Y){initialize();}
	Matrix_multiplication(const Matrix_multiplication &copied_matrix_mult){
		ROW_X = copied_matrix_mult.ROW_X;
		COL_X = copied_matrix_mult.COL_X;
		ROW_Y = copied_matrix_mult.ROW_Y;
		COL_Y = copied_matrix_mult.COL_Y;

		h_X = new DATA_TYPE[ROW_X * COL_X];
		h_Y = new DATA_TYPE[ROW_Y * COL_Y];
		h_Z = new DATA_TYPE[ROW_X * COL_Y];

		for (int i = 0; i < ROW_X * COL_X; i++){h_X[i] = copied_matrix_mult.h_X[i];}
		for (int i = 0; i < ROW_Y * COL_Y; i++){h_Y[i] = copied_matrix_mult.h_Y[i];}
		for (int i = 0; i < ROW_X * COL_Y; i++){h_Z[i] = copied_matrix_mult.h_Z[i];}
	}
	~Matrix_multiplication(){	
		delete [] h_X;
		delete [] h_Y;
		delete [] h_Z;

		hipFree(d_X);
		hipFree(d_Y);
		hipFree(d_Z);
	}

	void initialize(){
		h_X = new DATA_TYPE[ROW_X * COL_X];
		h_Y = new DATA_TYPE[ROW_Y * COL_Y];
		h_Z = new DATA_TYPE[ROW_X * COL_Y];
	}

	DATA_TYPE *d_X;
	DATA_TYPE *d_Y;
	DATA_TYPE *d_Z;
	DATA_TYPE *h_X;
	DATA_TYPE *h_Y; 
	DATA_TYPE *h_Z;

	int ROW_X;
	int COL_X;
	int ROW_Y; 
	int COL_Y;
} Matrix_multiplication;

// GPU KERNEL NAIVE KERNEL
__global__
void naive_matrix_multiplication
(
	DATA_TYPE * d_X,
 	DATA_TYPE * d_Y,
 	DATA_TYPE * d_Z,
 	unsigned int width_X,
 	unsigned int width_Z, 
 	unsigned int length_Z
 ){
	// thread id in thread block
	int local_th_id = threadIdx.x;
	int global_th_id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(global_th_id >= length_Z)
		return;

	DATA_TYPE temp = 0;
	int index_X = width_X * (global_th_id / width_Z);
	int index_Y = global_th_id % width_Z;

	for (int i = 0; i < width_X; i++){
		temp += d_X[index_X] * d_Y[index_Y];
		index_X++;
		index_Y += width_Z;
	}

	d_Z[global_th_id] = temp;
}

// GPU TILED KERNEL
__global__
void tiled_matrix_multiplication
(
	DATA_TYPE *d_X,
	DATA_TYPE *d_Y,
	DATA_TYPE *d_Z,
	int width_X,
	int width_Z,
	int height_Z
){
	__shared__ DATA_TYPE X_tile[WIDTH_TILE][WIDTH_TILE];
	__shared__ DATA_TYPE Y_tile[WIDTH_TILE][WIDTH_TILE];
	
	int t_row = threadIdx.y; int t_col = threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y; int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	DATA_TYPE acc_result = 0;

	for (int i = 0; i < (width_X + WIDTH_TILE - 1) / WIDTH_TILE; i++){
		if (row < height_Z && (i*WIDTH_TILE + t_col) < width_X)
			X_tile[t_row][t_col] = d_X[row * width_X + i * WIDTH_TILE + t_col];
		else
			X_tile[t_row][t_col] = 0;

		if (col < width_Z && (i*WIDTH_TILE + t_row) < width_X)
			Y_tile[t_row][t_col] = d_Y[(t_row + WIDTH_TILE*i)*width_Z + col];
		else
			Y_tile[t_row][t_col] = 0;

		__syncthreads();
		
		for (int k = 0; k < WIDTH_TILE; k++){
			acc_result += X_tile[t_row][k] * Y_tile[k][t_col];
		}
		__syncthreads();
	}

	if (row < height_Z && col < width_Z)
		d_Z[row * width_Z + col] = acc_result;
}

// TRANSFER MATRIX DATA TO DEVICE
void matrix_data_transfer_to_device(Matrix_multiplication* matrix_mult){

	const int ROW_X = matrix_mult->ROW_X;
	const int COL_X = matrix_mult->COL_X;
	const int ROW_Y = matrix_mult->ROW_Y;
	const int COL_Y = matrix_mult->COL_Y;

	// ALLOCATE MEMORY TO DEVICE
	hipMalloc((void**) &(matrix_mult->d_X), sizeof(DATA_TYPE) * ROW_X * COL_X);
	hipMalloc((void**) &(matrix_mult->d_Y), sizeof(DATA_TYPE) * ROW_Y * COL_Y);
	hipMalloc((void**) &(matrix_mult->d_Z), sizeof(DATA_TYPE) * ROW_X * COL_Y);

	// COPY MEMORY FROM HOST TO DEVICE
	hipMemcpy(matrix_mult->d_X, matrix_mult->h_X, sizeof(DATA_TYPE) * ROW_X * COL_X, hipMemcpyDefault);
	hipMemcpy(matrix_mult->d_Y, matrix_mult->h_Y, sizeof(DATA_TYPE) * ROW_Y * COL_Y, hipMemcpyDefault);
	hipMemcpy(matrix_mult->d_Z, matrix_mult->h_Z, sizeof(DATA_TYPE) * ROW_X * COL_Y, hipMemcpyDefault);
}

// EXECUTE NAIVE MATRIX MULTIPLICATION KERNEL 
void call_naive_matrix_multiplication_kernel(Matrix_multiplication* matrix_mult){
	const int THREADS_PER_BLOCK = 1024;
	const int THREAD_BLOCKS = (matrix_mult->ROW_X * matrix_mult->COL_Y + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

	naive_matrix_multiplication<<<THREAD_BLOCKS, THREADS_PER_BLOCK>>>(matrix_mult->d_X, matrix_mult->d_Y, matrix_mult->d_Z, matrix_mult->COL_X, matrix_mult->COL_Y, matrix_mult->ROW_X * matrix_mult->COL_Y);
	hipDeviceSynchronize();
}

// EXECUTE TILED MATRIX MULTIPLICATION KERNEL 
void call_tiled_matrix_multiplication_kernel(Matrix_multiplication* matrix_mult){
	dim3 blocks((matrix_mult->COL_Y + WIDTH_TILE - 1) /WIDTH_TILE,(matrix_mult->ROW_X + WIDTH_TILE - 1) /WIDTH_TILE);
	dim3 threads(WIDTH_TILE, WIDTH_TILE);

	tiled_matrix_multiplication<<<blocks, threads>>>(matrix_mult->d_X, matrix_mult->d_Y, matrix_mult->d_Z, matrix_mult->COL_X, matrix_mult->COL_Y, matrix_mult->ROW_X);
	hipDeviceSynchronize();
}

// COMPARE TWO MATRIX MULTIPLICATION RESULT 
bool compare_multiplication_result(DATA_TYPE matrix_result1 [], DATA_TYPE matrix_result2[], int result_length){

	for (int i = 0; i < result_length; i++){
		if (matrix_result1[i] != matrix_result2[i]){
			return false;
		}
	}

	return true;
}

// TEST MATRIX MULTIPLICATION KERNEL
void check_matrix_multiplication_result(const int trial_num){
	unsigned int seed = chrono::system_clock::now().time_since_epoch().count();
	mt19937 gen(seed);
	uniform_int_distribution<mt19937::result_type> dist_for_element(1,50);
	uniform_int_distribution<mt19937::result_type> dist_for_dim(100, 500);

	for(int trial_count = 0; trial_count < trial_num; trial_count++){
		int ROW_X, COL_X, ROW_Y, COL_Y;
		
		// INITIALIZE X, Y, Z MATRIX DIMENSION		
		ROW_X = dist_for_dim(gen);
		COL_X = ROW_Y = dist_for_dim(gen);
		COL_Y = dist_for_dim(gen);
		cout << "ROUND " << trial_count + 1 << ": TEST MATRIX MULTIPLICATION... X(" << ROW_X << "," << COL_X << ") and Y(" << ROW_Y << "," << COL_Y << ")" << endl;
		// MATRIX MULTIPLICATION STRUCT FOR NAIVE KERNEL
		Matrix_multiplication *test_matrix_mult = new Matrix_multiplication(ROW_X, COL_X, ROW_Y, COL_Y);

		// INITIALIZATION TWO MATRIX
		for (int i = 0; i < ROW_X * COL_X; i++){ test_matrix_mult->h_X[i] = DATA_TYPE(dist_for_element(gen));}
		for (int i = 0; i < ROW_Y * COL_Y; i++){ test_matrix_mult->h_Y[i] = DATA_TYPE(dist_for_element(gen));}
		
		// MATRIX MULTIPLICATION STRUCT FOR TILED KERNEL
		Matrix_multiplication *test_matrix_mult2 = new Matrix_multiplication(*test_matrix_mult);

		// DATA TRANSFER FROM HOST TO DEVICE
		matrix_data_transfer_to_device(test_matrix_mult);
		matrix_data_transfer_to_device(test_matrix_mult2);

		// KERNEL EXECUTION
		call_naive_matrix_multiplication_kernel(test_matrix_mult);
		call_tiled_matrix_multiplication_kernel(test_matrix_mult2);

		// DATA TRANSFER FROM DEIVE TO HOST 
		hipMemcpy(test_matrix_mult->h_Z, test_matrix_mult->d_Z, sizeof(DATA_TYPE) * ROW_X * COL_Y, hipMemcpyDefault);
		hipMemcpy(test_matrix_mult2->h_Z, test_matrix_mult2->d_Z, sizeof(DATA_TYPE) * ROW_X * COL_Y, hipMemcpyDefault);

		// EIGEN MATRIX MULTIPLICATION FOR TESTING NAIVE MATRIX MULTIPLICATION KERNEL  
		Matrix<int,-1,-1,RowMajor> matrix_dense_X, matrix_dense_Y, matrix_dense_Z; 
		// CONVERT 1D ARRAY TO EIGEN DENSE MATRIX  
		matrix_dense_X = Map<Matrix<int,-1,-1,RowMajor>, 0, OuterStride<>>(test_matrix_mult->h_X, ROW_X, COL_X, OuterStride<>(COL_X));
		matrix_dense_Y = Map<Matrix<int,-1,-1,RowMajor>, 0, OuterStride<>>(test_matrix_mult->h_Y, ROW_Y, COL_Y, OuterStride<>(COL_Y));
		// EIGEN MATRIX MULTIPLICATION AND GET 1D RESULT ARRAY
		matrix_dense_Z = matrix_dense_X * matrix_dense_Y;
		DATA_TYPE *eigen_matrix_result = matrix_dense_Z.data();

		// COMPARE MULTIPLICATION RESULT
		bool compare_result_naive_and_eigen;
		bool compare_result_tiled_and_naive;

		compare_result_naive_and_eigen = compare_multiplication_result(test_matrix_mult->h_Z, eigen_matrix_result, ROW_X*COL_Y);
		if (compare_result_tiled_and_naive)
			cout << "Eigen vs naive => " << "same matrix multiplication result." << endl;

		compare_result_tiled_and_naive = compare_multiplication_result(test_matrix_mult->h_Z, test_matrix_mult2->h_Z, ROW_X*COL_Y);
		if (compare_result_tiled_and_naive)
			cout << "Tiled vs naive => " << "same matrix multiplication result." << endl;

		// DELETE TESTED MATRIX
		delete test_matrix_mult;
		delete test_matrix_mult2;
	}
}

// PRINT 2D MATRIX 
void print_matrix(DATA_TYPE matrix [], int ROW, int COL){
	for (int i = 0; i < ROW * COL; i++){
		cout << matrix[i] << "  ";
		if((i+1)%COL==0)
			cout << "\n";
	}
}

int main (){

	std::chrono::milliseconds naive_matrix_multiplication_kernel_execution_time {};
	std::chrono::milliseconds tiled_matrix_multiplication_kernel_execution_time {};

	std::chrono::time_point<std::chrono::system_clock> matrix_multiplication_kernel_start_time;
	// MATRIX STRUCT
	Matrix_multiplication *matrix_mult = new Matrix_multiplication();

	const int ROW_X = matrix_mult->ROW_X;
	const int COL_X = matrix_mult->COL_X;
	const int ROW_Y = matrix_mult->ROW_Y;
	const int COL_Y = matrix_mult->COL_Y;
	
	// INITIALIZATION TWO MATRIX
	for (int i = 0; i < ROW_X * COL_X; i++){ matrix_mult->h_X[i] = DATA_TYPE(1);}
	for (int i = 0; i < ROW_Y * COL_Y; i++){ matrix_mult->h_Y[i] = DATA_TYPE(1);}

	// TRANSFER DATA IN MATRIX MULTIPLICATION TO DEVICE 
	matrix_data_transfer_to_device(matrix_mult);
	
	// CALL NAIVE MATRIX MULTIPLICATION KERNEL
	matrix_multiplication_kernel_start_time = std::chrono::system_clock::now();
	call_naive_matrix_multiplication_kernel(matrix_mult);
	naive_matrix_multiplication_kernel_execution_time = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now() - matrix_multiplication_kernel_start_time);
		
	// COPY RESULT MATRIX FROM DEVICE TO HOST
	hipMemcpy(matrix_mult->h_Z, matrix_mult->d_Z, sizeof(DATA_TYPE) * ROW_X * COL_Y, hipMemcpyDefault);

	// PRINT EXECUTION TIME AND MATRIX
	cout << "Naive Matrix multiplication kernel execution time : " << naive_matrix_multiplication_kernel_execution_time.count() << "ms" << endl;

	// CALL TILED MATRIX MULIPLICATION KERNEL 
	matrix_multiplication_kernel_start_time = std::chrono::system_clock::now();
	call_tiled_matrix_multiplication_kernel(matrix_mult);
	tiled_matrix_multiplication_kernel_execution_time = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now() - matrix_multiplication_kernel_start_time);

	// COPY RESULT MATRIX FROM DEVICE TO HOST
	hipMemcpy(matrix_mult->h_Z, matrix_mult->d_Z, sizeof(DATA_TYPE) * ROW_X * COL_Y, hipMemcpyDefault);

	// PRINT EXECUTION TIME AND MATRIX
	cout << "Tiled Matrix multiplication kernel execution time : " << tiled_matrix_multiplication_kernel_execution_time.count() << "ms" << endl;

	delete matrix_mult;
	return 0;
}
